#include "mylib.h"
#include "mylib.cuh"
#include "opencv2/opencv.hpp"
#include <hip/hip_runtime.h>


int main(int, char**)
{ 	
	VideoCapture cap(0); // open the default camera
	if(!cap.isOpened())  // check if we succeeded
		return -1;
	
	while(1){
		Mat frame;
	    	cap >> frame;
	    	char c=(char)waitKey(25);
		if(c == '1'){				// if '1' est appuye
			Mat NB = plus_proche_voisin(frame,2);
	    		imshow("plus proche voisin", NB);
			}
		else if(c == '0') destroyAllWindows();	// if '0' est appuye

		else imshow("frame", frame);
			
	    	if(c==27)				// if 'esc' est appuye
	      		break;
	}
  	// When everything done, release the video capture object
	cap.release();

	// Closes all the frames
	destroyAllWindows();

	return 0;
}
