#include "mylib.h"
#include "mylib.cuh"
#include <hip/hip_runtime.h>

// acces au flux de la camera
std::string gstreamer_pipeline (int capture_width, int capture_height, int display_width, int display_height, int framerate, int flip_method) {
    return "nvarguscamerasrc ! video/x-raw(memory:NVMM), width=(int)" + std::to_string(capture_width) + ", height=(int)" +
           std::to_string(capture_height) + ", format=(string)NV12, framerate=(fraction)" + std::to_string(framerate) +
           "/1 ! nvvidconv flip-method=" + std::to_string(flip_method) + " ! video/x-raw, width=(int)" + std::to_string(display_width) + ", height=(int)" +
           std::to_string(display_height) + ", format=(string)BGRx ! videoconvert ! video/x-raw, format=(string)BGR ! appsink";
}

int main(int, char**)
{ 
    	int capture_width = 640;
    	int capture_height = 360 ;
    	int display_width = 640 ;
    	int display_height = 360 ;
    	int framerate = 60 ;
    	int flip_method = 0 ;
		int resolution = 2;

    	std::string pipeline = gstreamer_pipeline(capture_width,
		capture_height,
		display_width,
		display_height,
		framerate,
		flip_method);
    	std::cout << "Using pipeline: \n\t" << pipeline << "\n";

	cv::VideoCapture cap(pipeline, cv::CAP_GSTREAMER);
	//VideoCapture cap(); // open the default camera
	if(!cap.isOpened())  // check if we succeeded
		return -1;
	
	while(1){
		Mat frame;
	    	cap >> frame;
	    	char c=(char)waitKey(25);

		if(c == '1'){				// if '1' est appuye
			Mat PPV = plus_proche_voisin(frame,resolution);
	    		imshow("plus proche voisin", PPV);
		}

		if(c == '2'){				// if '2' est appuye
			Mat BL = bilineaire(frame,resolution);
	    		imshow("bilineaire", BL);
		}

		if( c== '3'){				// if '3' est appuye
			Mat BC = bicubique(frame,resolution);
	    		imshow("bicubique", BC);
		}
				if( c== '4'){				// if '4' est appuye
			Mat BNC =PPV_GPU(frame,resolution);
	    		imshow("PPV_GPU", BNC);
		}
				if( c== '5'){				// if '5' est appuye
			Mat Bilig =bili_GPU(frame,resolution);
	    		imshow("bili_GPU", Bilig);
		}
				if( c== '6'){				// if '5' est appuye
			Mat Bicub =bicubic_GPU(frame,resolution);
	    		imshow("bicubic_GPU", Bicub);
		}
				if( c== '7'){				// if '5' est appuye
			Mat echan =echantillonnage(frame,resolution);
	    		imshow("echantillonnage", echan);
		}
		
		
		if(c == '8') destroyAllWindows();	// if '6' est appuye

		else imshow("frame", frame);
			
		       if(c==27)				// if 'esc' est appuye
	      		break;
	}
  	// When everything done, release the video capture object
	cap.release();

	// Closes all the frames
	destroyAllWindows();

	return 0;
}
