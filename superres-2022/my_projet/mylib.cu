#include "hip/hip_runtime.h"
#include "mylib.h"
#include "mylib.cuh"
#include <iostream>
#include <chrono>

__global__ void kernel_PPV_GPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j,int res)
{
	int i,j,vi,vj;
	

	int height = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	int width = blockIdx.y*BLOCK_SIZE;

	i = height + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = width + threadIdx.y;

	for(vi=0;vi<res;vi++) // Parcourir les nouveaux pixels sur la même ligne
	{
		for(vj=0;vj<res;vj++)  // Parcourir les nouveaux pixels sur la même colonne
		{
			d_image_out[0+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = d_image_in[0+j*3+i*3*size_j];
			d_image_out[1+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = d_image_in[1+j*3+i*3*size_j];
			d_image_out[2+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = d_image_in[2+j*3+i*3*size_j];
		}	
	}
}

Mat PPV_GPU( Mat in,int res)
{
	hipError_t error;
	Mat out;
	out.create(in.rows*res,in.cols*res,CV_8UC3);
	auto start= std::chrono::high_resolution_clock::now();

	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size*res*res);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_PPV_GPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols,res);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size*res*res,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	auto end= std::chrono::high_resolution_clock::now();
   auto resultat=std::chrono::duration_cast<std::chrono::milliseconds>(end-start);
   std::cout<< "Temps Plus Proche Voisin avec GPU ="<<resultat.count()<<"millisecondes "<<endl;
	return out;
}

__global__ void kernel_bili_GPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j,int res)
{
	int i,j,vi,vj,P,X=0,N=0, coef;
	int a ,b ;
	int tab_ref[5][2];
	int diff_rows,diff_cols;
	unsigned char *d_image_out_bis;
	
	int height = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	int width = blockIdx.y*BLOCK_SIZE;

	i = height + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = width + threadIdx.y;

		//echantillonage
		d_image_out[1+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[1+j*3+i*3*size_j];
		d_image_out[0+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[0+j*3+i*3*size_j];
		d_image_out[2+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[2+j*3+i*3*size_j];
	
	for(vi=0;vi<res;vi++) // Parcourir les nouveaux pixels sur la même ligne
	{
		for(vj=0;vj<res;vj++) // Parcourir les nouveaux pixels sur la même colonne
		{
			if((vi>0) || (vj>0)) // Si pixel inconnu
			{
				a=1;
			}		
			else if((vi>0) && (vj>0)) // Si pixel connu
			{
				a=0;
			}
			d_image_out[0+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = (d_image_in[0+j*3+i*3*size_j]+d_image_in[0+(j+1)*3+(i+1)*3*size_j]+d_image_in[0+(j+1)*3+i*3*size_j]+d_image_in[0+j*3+(i+1)*3*size_j])/(4);
			d_image_out[1+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = (d_image_in[1+j*3+i*3*size_j]+d_image_in[1+(j+1)*3+(i+1)*3*size_j]+d_image_in[1+(j+1)*3+i*3*size_j]+d_image_in[1+j*3+(i+1)*3*size_j])/(4);
			d_image_out[2+3*(j*res+vj)+(i*res+vi)*3*size_j*res] = (d_image_in[2+j*3+i*3*size_j]+d_image_in[2+(j+1)*3+(i+1)*3*size_j]+d_image_in[2+(j+1)*3+i*3*size_j]+d_image_in[2+j*3+(i+1)*3*size_j])/(4);
		}
	}		
}

Mat bili_GPU( Mat in,int res)
{
	hipError_t error;
	Mat out;
	out.create(in.rows*res,in.cols*res,CV_8UC3);
	
	auto start= std::chrono::high_resolution_clock::now();
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	/*hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);*/
	
	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size*res*res);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	//error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	//error = hipEventSynchronize(stop_mem);
	//float msecMem = 0.0f;
	//error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_bili_GPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols,res);
	
	// Record the start event
	//error = hipEventRecord(start_mem, NULL);
	//error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size*res*res,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	/*
	float msecTotal,msecMem2;
	error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);
	error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	*/
	auto end= std::chrono::high_resolution_clock::now();
   auto resultat=std::chrono::duration_cast<std::chrono::milliseconds>(end-start);
   std::cout<< "Temps Bilinéaire avec GPU ="<<resultat.count()<<"millisecondes "<<endl;
   
	return out;
}

__global__ void kernel_bicubic_GPU(unsigned char *d_image_in, unsigned char *d_image_out,int size_j,int res)
{
	int i,j,vi,vj,P,X=0,N=0, coef;
	int a ,b ,M,Q,tab_ref1[4];
	int tab_ref[5][2], rseu[4][3];
	int diff_rows,diff_cols;
	unsigned char *d_image_out_bis;
	
	int height = blockIdx.x*BLOCK_SIZE; // num de block dans la grille de block
	int width = blockIdx.y*BLOCK_SIZE;

	i = height + threadIdx.x;// recuperer l'identifiant d'un thread dans les blocs
	j = width + threadIdx.y;

		//echantillonage
		d_image_out[1+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[1+j*3+i*3*size_j];
		d_image_out[0+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[0+j*3+i*3*size_j];
		d_image_out[2+(j*res)*3+(i*res*res)*3*size_j] = d_image_in[2+j*3+i*3*size_j];
	
	for(vi=0;vi<res;vi++) // Parcourir les nouveaux pixels sur la même ligne
	{
		for(vj=0;vj<res;vj++) // Parcourir les nouveaux pixels sur la même colonne
		{
			if ((vi<=0) && (vj<=0))
			{
				
			}
			else if(vi==0) // Si pixel inconnu colonne
			{
				for(int M=0;M<4;M++)
				{
					tab_ref1[M]=i-2*(1-M);
				}
				for (int R=0;R<4;R++) 
				{
					float p=1.0;
					for(int T=0;T<4;T++)
					{
						if(R!=T)
						{
							p = p* (i - tab_ref1[T])/(tab_ref1[R] - tab_ref1[T]);
						}
					}
					for ( int A=0;A<3;A++)
					{
						Q=round(p * d_image_in[A+j*3+tab_ref1[M]*3*size_j]);
						d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res]= d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res] + Q ;
					} 
				}
			}		
			else if(vj==0) // Si pixel connu
			{
				for(int M=0;M<4;M++)
				{
					tab_ref1[M]=j-2*(1-M);
				}
				for (int R=0;R<4;R++) 
				{
					float p=1.0;
					for(int T=0;T<4;T++)
					{
						if(R!=T)
						{
							p = p* (j - tab_ref1[T])/(tab_ref1[R] - tab_ref1[T]);
						}
					}
					for ( int A=0;A<3;A++)
					{
						Q=round(p * d_image_in[A+j*3+tab_ref1[M]*3*size_j]);
						d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res]= d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res] + Q ;
					} 
				}
			}
			
			else 
			{
				for (int Y=0;Y<4;Y++)
				{
					for(int M=0;M<4;M++)
					{
						tab_ref1[M]=j-2*(1-M);
					}
					for (int R=0;R<4;R++) 
					{
						float p=1.0;
						for(int T=0;T<4;T++)
						{
							if(R!=T)
							{
								p = p* (j - tab_ref1[T])/(tab_ref1[R] - tab_ref1[T]);
							}
						}
						for ( int A=0;A<3;A++)
						{
							Q=round(p * d_image_in[A+j-(1-Y)*3+tab_ref1[M]*3*size_j]);
							rseu[Y][A]= rseu[Y][A] + Q ;
						} 
					}
				}
				for(int M=0;M<4;M++)
				{
					tab_ref1[M]=i-2*(1-M);
				}
				for (int R=0;R<4;R++) 
				{
					float p=1.0;
					for(int T=0;T<4;T++)
					{
						if(R!=T)
						{
							p = p* (i - tab_ref1[T])/(tab_ref1[R] - tab_ref1[T]);
						}
					}
					for ( int A=0;A<3;A++)
					{
						Q=round(p * rseu[M][A]);
						d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res]= d_image_out[A+3*(j*res+vj)+(i*res+vi)*3*size_j*res] + Q ;
					} 
				}
			}
		}
	}		
}

Mat bicubic_GPU( Mat in,int res)
{
	hipError_t error;
	Mat out;
	float msecTotal,msecMem2,msecMem = 0.0f;;
	out.create(in.rows*res,in.cols*res,CV_8UC3);
	
	//auto start= std::chrono::high_resolution_clock::now();
	// allocate host memory
	unsigned char *h_image_in_GPU ;
	h_image_in_GPU=in.data;
	
	hipEvent_t start,stop,start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);
	
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);
	

	// images on device memoryÍÍÍ
	unsigned char *d_image_in_GPU;
	unsigned char *d_image_out_GPU;
	
	const unsigned long int mem_size=in.cols*in.rows*3*sizeof(unsigned char);
	
	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc((void**) &d_image_in_GPU,mem_size );
	hipMalloc((void**) &d_image_out_GPU, mem_size*res*res);
	
	// copy host memory to device
	hipMemcpy(d_image_in_GPU, h_image_in_GPU,mem_size ,hipMemcpyHostToDevice);
	
	error = hipEventRecord(stop_mem, NULL);
	
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop_mem);
	error = hipEventElapsedTime(&msecMem, start, stop_mem);
	
	// setup execution parameters -> découpage en threads
	dim3 threads(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(in.rows/BLOCK_SIZE,in.cols/BLOCK_SIZE);
	
	// lancement des threads executé sur la carte GPU
	kernel_bicubic_GPU<<< grid, threads >>>(d_image_in_GPU, d_image_out_GPU,in.cols,res);
	
	// Record the start event
	error = hipEventRecord(start_mem, NULL);
	error = hipEventSynchronize(start_mem);
	
	// copy result from device to host
	hipMemcpy(out.data, d_image_out_GPU, mem_size*res*res,hipMemcpyDeviceToHost);
	hipFree(d_image_in_GPU);
	hipFree(d_image_out_GPU);
	
	
	/*error = hipEventRecord(stop, NULL);
	error = hipEventSynchronize(stop);*/
	/*error = hipEventElapsedTime(&msecTotal, start, stop);
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);*/
	
	/*auto end= std::chrono::high_resolution_clock::now();
   auto resultat=std::chrono::duration_cast<std::chrono::milliseconds>(end-start);
   std::cout<< "Temps Bicubique avec GPU ="<<resultat.count()<<"millisecondes "<<endl;*/
  //  std::cout<< "Temps Bicubique avec GPU ="<<msecTotal<<"millisecondes "<<"calcul "<<msecTotal-(msecMem+msecMem2)<<"("<<(1-(msecMem+msecMem2)/msecTotal)*100<<")"<<endl;
	return out;
}